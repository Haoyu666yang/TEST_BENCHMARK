#include <iostream>
#include <vector>
#include <string>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include "binary_read.h"
#include "measure_sort_time.h"
#include "write_csv.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(x)); \
    exit(EXIT_FAILURE);}} while(0)

template <typename T>
void cub_sort(std::vector<T>& data) {
    T *d_input, *d_output;
    size_t num_items = data.size();
    CUDA_CALL(hipMalloc(&d_input, num_items * sizeof(T)));
    CUDA_CALL(hipMalloc(&d_output, num_items * sizeof(T)));
    CUDA_CALL(hipMemcpy(d_input, data.data(), num_items * sizeof(T), hipMemcpyHostToDevice));

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_input, d_output, num_items));
    CUDA_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_input, d_output, num_items));

    CUDA_CALL(hipMemcpy(data.data(), d_output, num_items * sizeof(T), hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_input));
    CUDA_CALL(hipFree(d_output));
    CUDA_CALL(hipFree(d_temp_storage));
}

void cub_sort_int32(std::vector<int32_t>& data) {
    cub_sort(data);
}

void cub_sort_int64(std::vector<int64_t>& data) {
    cub_sort(data);
}

int main() {
    int runs = 20;
    std::vector<int> sizes = {8, 11, 14, 17};

    for (int size : sizes) {
        std::vector<int32_t> uniform_data_int32;
        std::vector<int32_t> normal_data_int32;
        std::vector<int32_t> zipf_data_int32;

        std::vector<int64_t> uniform_data_int64;
        std::vector<int64_t> normal_data_int64;
        std::vector<int64_t> zipf_data_int64;

        std::string size_str = std::to_string(1 << size);

        if (!binary_read_file("origin_data/uniform_data_int32_size_" + size_str + ".bin", uniform_data_int32)) {
            std::cerr << "error opening file: origin_data/uniform_data_int32_size_" + size_str + ".bin" << std::endl;
            continue;
        }
        if (!binary_read_file("origin_data/normal_data_int32_size_" + size_str + ".bin", normal_data_int32)) {
            std::cerr << "error opening file: origin_data/normal_data_int32_size_" + size_str + ".bin" << std::endl;
            continue;
        }
        if (!binary_read_file("origin_data/zipf_data_int32_size_" + size_str + ".bin", zipf_data_int32)) {
            std::cerr << "error opening file: origin_data/zipf_data_int32_size_" + size_str + ".bin" << std::endl;
            continue;
        }

        if (!binary_read_file("origin_data/uniform_data_int64_size_" + size_str + ".bin", uniform_data_int64)) {
            std::cerr << "error opening file: origin_data/uniform_data_int64_size_" + size_str + ".bin" << std::endl;
            continue;
        }
        if (!binary_read_file("origin_data/normal_data_int64_size_" + size_str + ".bin", normal_data_int64)) {
            std::cerr << "error opening file: origin_data/normal_data_int64_size_" + size_str + ".bin" << std::endl;
            continue;
        }
        if (!binary_read_file("origin_data/zipf_data_int64_size_" + size_str + ".bin", zipf_data_int64)) {
            std::cerr << "error opening file: origin_data/zipf_data_int64_size_" + size_str + ".bin" << std::endl;
            continue;
        }

        double avg_time_uniform_int32 = measure_sort_time(cub_sort_int32, uniform_data_int32, runs);
        double avg_time_normal_int32 = measure_sort_time(cub_sort_int32, normal_data_int32, runs);
        double avg_time_zipf_int32 = measure_sort_time(cub_sort_int32, zipf_data_int32, runs);

        double avg_time_uniform_int64 = measure_sort_time(cub_sort_int64, uniform_data_int64, runs);
        double avg_time_normal_int64 = measure_sort_time(cub_sort_int64, normal_data_int64, runs);
        double avg_time_zipf_int64 = measure_sort_time(cub_sort_int64, zipf_data_int64, runs);

        std::vector<std::pair<std::string, double>> int32_results = {
            {"Uniform", avg_time_uniform_int32},
            {"Normal", avg_time_normal_int32},
            {"Zipf", avg_time_zipf_int32}
        };
        write_csv("result_time/int32_cub_sort_times_size_" + size_str + ".csv", int32_results);

        std::vector<std::pair<std::string, double>> int64_results = {
            {"Uniform", avg_time_uniform_int64},
            {"Normal", avg_time_normal_int64},
            {"Zipf", avg_time_zipf_int64}
        };
        write_csv("result_time/int64_cub_sort_times_size_" + size_str + ".csv", int64_results);
    }

    return 0;
}


// #include <iostream>
// #include <vector>
// #include <string>
// #include <thrust/sort.h>
// #include <thrust/device_vector.h>
// #include <hip/hip_runtime.h> 
// #include "binary_read.h"
// #include "measure_sort_time.h"
// #include "write_csv.h"

// void thrust_sort_int32(std::vector<int32_t>& data) {
//     thrust::device_vector<int32_t> d_data = data;  
//     hipDeviceSynchronize(); 
//     thrust::sort(d_data.begin(), d_data.end());    
//     hipDeviceSynchronize();
//     thrust::copy(d_data.begin(), d_data.end(), data.begin()); 
//     hipDeviceSynchronize(); 
// }

// void thrust_sort_int64(std::vector<int64_t>& data) {
//     thrust::device_vector<int64_t> d_data = data;  
//     hipDeviceSynchronize(); 
//     thrust::sort(d_data.begin(), d_data.end());   
//     hipDeviceSynchronize(); 
//     thrust::copy(d_data.begin(), d_data.end(), data.begin());  
//     hipDeviceSynchronize(); 
// }

// int main() {
//     int runs = 20;
//     std::vector<int> sizes = {8, 11, 14, 17};  

//     for (int size : sizes) {
//         std::vector<int32_t> uniform_data_int32;
//         std::vector<int32_t> normal_data_int32;
//         std::vector<int32_t> zipf_data_int32;

//         std::vector<int64_t> uniform_data_int64;
//         std::vector<int64_t> normal_data_int64;
//         std::vector<int64_t> zipf_data_int64;

//         std::string size_str = std::to_string(1 << size);  

//         if (!binary_read_file("origin_data/uniform_data_int32_size_" + size_str + ".bin", uniform_data_int32)) {
//             std::cerr << "error opening file: origin_data/uniform_data_int32_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }
//         if (!binary_read_file("origin_data/normal_data_int32_size_" + size_str + ".bin", normal_data_int32)) {
//             std::cerr << "error opening file: origin_data/normal_data_int32_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }
//         if (!binary_read_file("origin_data/zipf_data_int32_size_" + size_str + ".bin", zipf_data_int32)) {
//             std::cerr << "error opening file: origin_data/zipf_data_int32_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }

//         if (!binary_read_file("origin_data/uniform_data_int64_size_" + size_str + ".bin", uniform_data_int64)) {
//             std::cerr << "error opening file: origin_data/uniform_data_int64_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }
//         if (!binary_read_file("origin_data/normal_data_int64_size_" + size_str + ".bin", normal_data_int64)) {
//             std::cerr << "error opening file: origin_data/normal_data_int64_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }
//         if (!binary_read_file("origin_data/zipf_data_int64_size_" + size_str + ".bin", zipf_data_int64)) {
//             std::cerr << "error opening file: origin_data/zipf_data_int64_size_" + size_str + ".bin" << std::endl;
//             continue;
//         }

//         double avg_time_uniform_int32 = measure_sort_time(thrust_sort_int32, uniform_data_int32, runs);
//         double avg_time_normal_int32 = measure_sort_time(thrust_sort_int32, normal_data_int32, runs);
//         double avg_time_zipf_int32 = measure_sort_time(thrust_sort_int32, zipf_data_int32, runs);

//         double avg_time_uniform_int64 = measure_sort_time(thrust_sort_int64, uniform_data_int64, runs);
//         double avg_time_normal_int64 = measure_sort_time(thrust_sort_int64, normal_data_int64, runs);
//         double avg_time_zipf_int64 = measure_sort_time(thrust_sort_int64, zipf_data_int64, runs);

//         std::vector<std::pair<std::string, double>> int32_results = {
//             {"Uniform", avg_time_uniform_int32},
//             {"Normal", avg_time_normal_int32},
//             {"Zipf", avg_time_zipf_int32}
//         };
//         write_csv("result_time/int32_thrust_sort_times_size_" + size_str + ".csv", int32_results);

//         std::vector<std::pair<std::string, double>> int64_results = {
//             {"Uniform", avg_time_uniform_int64},
//             {"Normal", avg_time_normal_int64},
//             {"Zipf", avg_time_zipf_int64}
//         };
//         write_csv("result_time/int64_thrust_sort_times_size_" + size_str + ".csv", int64_results);

//     }

//     return 0;
// }
